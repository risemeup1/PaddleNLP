// Copyright (c) 2024 PaddlePaddle Authors. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
#pragma once

#include "mla_cache_kernel.cuh"

template <paddle::DataType T>
std::vector<paddle::Tensor> PrefillMLAWriteCache(
                    const AppendAttnMetaData& meta_data,
                    const paddle::Tensor& kv_nope,
                    const paddle::Tensor& kv_pe,
                    const paddle::Tensor& seq_lens,
                    const paddle::Tensor& seq_lens_decoder,
                    const paddle::Tensor& padding_offsets,
                    const paddle::Tensor& cum_offsets,
                    const paddle::Tensor& block_tables,
                    const int max_seq_len,
                    hipStream_t& stream,
                    paddle::Tensor* kv_cache) {
  typedef PDTraits<T> traits_;
  typedef typename traits_::DataType DataType_;
  typedef typename traits_::data_t data_t;

  auto max_blocks_per_seq = meta_data.max_blocks_per_seq;
  auto num_tokens = meta_data.token_nums;
  auto block_size = meta_data.block_size;
  auto nope_size = meta_data.head_dims_v;
  auto all_size = meta_data.head_dims;
  int pe_size = all_size - nope_size;
  auto kv_num_heads = meta_data.kv_num_heads;
  const uint32_t elem_nums = num_tokens * kv_num_heads * all_size;

  constexpr int PackSize = 16 / sizeof(DataType_);
  const int pack_num = elem_nums / PackSize;
  const int blocksize = 128;
  int grid_size = 1;
  GetNumBlocks<128>(pack_num, &grid_size);

  prefill_absorb_cache_kernel<DataType_, PackSize>
      <<<grid_size, blocksize, 0, stream>>>(
          reinterpret_cast<DataType_*>(const_cast<data_t*>(kv_nope.data<data_t>())),
          reinterpret_cast<DataType_*>(const_cast<data_t*>(kv_pe.data<data_t>())),
          reinterpret_cast<DataType_*>(kv_cache->data<data_t>()),
          block_tables.data<int>(),
          padding_offsets.data<int>(),
          cum_offsets.data<int>(),
          seq_lens.data<int>(),
          seq_lens_decoder.data<int>(),
          max_seq_len,
          max_blocks_per_seq,
          kv_num_heads,
          nope_size,
          pe_size,
          block_size,
          elem_nums);
  return {};
}

std::vector<paddle::Tensor> PrefillMLAWriteCacheKernel(
    const paddle::Tensor& kv_nope,
    const paddle::Tensor& kv_pe,
    const paddle::Tensor& kv_cache,
    const paddle::Tensor& seq_lens,
    const paddle::Tensor& seq_lens_decoder,
    const paddle::Tensor& padding_offsets,
    const paddle::Tensor& cum_offsets,
    const paddle::Tensor& block_tables,
    const std::string& cache_quant_type_str,
    const int max_seq_len) {
  hipStream_t stream = kv_pe.stream();
  AppendAttnMetaData meta_data;
  const auto& kv_nope_dims = kv_nope.dims();
  const auto& kv_pe_dims = kv_pe.dims();
  const auto& kv_cache_dims = kv_cache.dims();
  meta_data.kv_num_heads = kv_cache_dims[1];
  const auto nope_size = kv_nope_dims[kv_nope_dims.size() - 1] / meta_data.kv_num_heads;
  meta_data.token_nums = kv_nope_dims[0];
  meta_data.head_dims = kv_cache_dims[3];
  meta_data.head_dims_v = nope_size;

  meta_data.max_blocks_per_seq = block_tables.dims()[1];
  meta_data.block_size = kv_cache_dims[2];
  meta_data.batch_size = cum_offsets.dims()[0];
  switch (kv_pe.dtype()) {
    case paddle::DataType::BFLOAT16: {
      return PrefillMLAWriteCache<paddle::DataType::BFLOAT16>(meta_data,
                              kv_nope,
                              kv_pe,
                              seq_lens,
                              seq_lens_decoder,
                              padding_offsets,
                              cum_offsets,
                              block_tables,
                              max_seq_len,
                              stream,
                              const_cast<paddle::Tensor*>(&kv_cache));
    }
    case paddle::DataType::FLOAT16: {
      return PrefillMLAWriteCache<paddle::DataType::FLOAT16>(meta_data,
                              kv_nope,
                              kv_pe,
                              seq_lens,
                              seq_lens_decoder,
                              padding_offsets,
                              cum_offsets,
                              block_tables,
                              max_seq_len,
                              stream,
                              const_cast<paddle::Tensor*>(&kv_cache));
    }
  }
  return {};
}

template <paddle::DataType T>
std::vector<paddle::Tensor> DecodeMLAWriteCache(
                    const AppendAttnMetaData& meta_data,
                    const paddle::Tensor& kv_nope,
                    const paddle::Tensor& kv_pe,
                    const paddle::Tensor& seq_lens,
                    const paddle::Tensor& seq_lens_encoder,
                    const paddle::Tensor& padding_offsets,
                    const paddle::Tensor& cum_offsets,
                    const paddle::Tensor& block_tables,
                    const int max_seq_len,
                    const bool speculate_decoder,
                    hipStream_t& stream,
                    paddle::Tensor* kv_cache) {
  typedef PDTraits<T> traits_;
  typedef typename traits_::DataType DataType_;
  typedef typename traits_::data_t data_t;
  
  auto max_blocks_per_seq = meta_data.max_blocks_per_seq;
  auto bsz = meta_data.batch_size;
  auto token_num = meta_data.token_nums;
  auto block_size = meta_data.block_size;
  auto nope_size = meta_data.head_dims_v;
  auto all_size = meta_data.head_dims;
  int pe_size = all_size - nope_size;
  auto kv_num_heads = meta_data.kv_num_heads;
  constexpr int PackSize = 16 / sizeof(DataType_);
  const int blocksize = 128;
  int grid_size = 1;


  if (speculate_decoder) {
    const uint32_t elem_nums = token_num * kv_num_heads * all_size;
    const int pack_num = elem_nums / PackSize;
    GetNumBlocks<128>(pack_num, &grid_size);
    speculate_decode_absorb_cache_kernel<DataType_, PackSize>
        <<<grid_size, blocksize, 0, stream>>>(
            reinterpret_cast<DataType_*>(const_cast<data_t*>(kv_nope.data<data_t>())),
            reinterpret_cast<DataType_*>(const_cast<data_t*>(kv_pe.data<data_t>())),
            reinterpret_cast<DataType_*>(kv_cache->data<data_t>()),
            block_tables.data<int>(),
            padding_offsets.data<int>(),
            cum_offsets.data<int>(),
            seq_lens.data<int>(),
            seq_lens_encoder.data<int>(),
            max_seq_len,
            max_blocks_per_seq,
            kv_num_heads,
            nope_size,
            pe_size,
            block_size,
            elem_nums);
  } else {
    const uint32_t elem_nums = bsz * kv_num_heads * all_size;
    const int pack_num = elem_nums / PackSize;
    GetNumBlocks<128>(pack_num, &grid_size);
    decode_absorb_cache_kernel<DataType_, PackSize>
        <<<grid_size, blocksize, 0, stream>>>(
            reinterpret_cast<DataType_*>(const_cast<data_t*>(kv_nope.data<data_t>())),
            reinterpret_cast<DataType_*>(const_cast<data_t*>(kv_pe.data<data_t>())),
            reinterpret_cast<DataType_*>(kv_cache->data<data_t>()),
            block_tables.data<int>(),
            cum_offsets.data<int>(),
            seq_lens.data<int>(),
            seq_lens_encoder.data<int>(),
            max_seq_len,
            max_blocks_per_seq,
            kv_num_heads,
            nope_size,
            pe_size,
            block_size,
            elem_nums);
  }
  return {};
}

std::vector<paddle::Tensor> DecodeMLAWriteCacheKernel(
    const paddle::Tensor& kv_nope,
    const paddle::Tensor& kv_pe,
    const paddle::Tensor& kv_cache,
    const paddle::Tensor& seq_lens,
    const paddle::Tensor& seq_lens_encoder,
    const paddle::Tensor& padding_offsets,
    const paddle::Tensor& cum_offsets,
    const paddle::Tensor& block_tables,
    const std::string& cache_quant_type_str,
    const int max_seq_len,
    const bool speculate_decoder) {
  hipStream_t stream = kv_pe.stream();
  AppendAttnMetaData meta_data;
  const auto& kv_nope_dims = kv_nope.dims();
  const auto& kv_pe_dims = kv_pe.dims();
  const auto& kv_cache_dims = kv_cache.dims();
  meta_data.kv_num_heads = kv_cache_dims[1];
  const auto nope_size = kv_nope_dims[kv_nope_dims.size() - 1] / meta_data.kv_num_heads;
  meta_data.token_nums = kv_nope_dims[0];
  meta_data.head_dims = kv_cache_dims[3];
  meta_data.head_dims_v = nope_size;

  meta_data.max_blocks_per_seq = block_tables.dims()[1];
  meta_data.block_size = kv_cache_dims[2];
  meta_data.batch_size = cum_offsets.dims()[0];
  switch (kv_pe.dtype()) {
    case paddle::DataType::BFLOAT16: {
      return DecodeMLAWriteCache<paddle::DataType::BFLOAT16>(meta_data,
                              kv_nope,
                              kv_pe,
                              seq_lens,
                              seq_lens_encoder,
                              padding_offsets,
                              cum_offsets,
                              block_tables,
                              max_seq_len,
                              speculate_decoder,
                              stream,
                              const_cast<paddle::Tensor*>(&kv_cache));
    }
    case paddle::DataType::FLOAT16: {
      return DecodeMLAWriteCache<paddle::DataType::FLOAT16>(meta_data,
                              kv_nope,
                              kv_pe,
                              seq_lens,
                              seq_lens_encoder,
                              padding_offsets,
                              cum_offsets,
                              block_tables,
                              max_seq_len,
                              speculate_decoder,
                              stream,
                              const_cast<paddle::Tensor*>(&kv_cache));
    }
  }
  return {};
}


PD_BUILD_OP(prefill_mla_write_cache)
    .Inputs({"kv_nope",
             "kv_pe",
             "kv_cache",
             "seq_lens",
             "seq_lens_decoder",
             "padding_offsets",
             "cum_offsets",
             "block_tables"})
    .Outputs({"kv_cache_out"})
    .SetInplaceMap({{"kv_cache", "kv_cache_out"}})
    .Attrs({"cache_quant_type_str: std::string",
            "max_seq_len: int"})
    .SetKernelFn(PD_KERNEL(PrefillMLAWriteCacheKernel));

PD_BUILD_OP(decode_mla_write_cache)
    .Inputs({"kv_nope",
             "kv_pe",
             "kv_cache",
             "seq_lens",
             "seq_lens_encoder",
             "padding_offsets",
             "cum_offsets",
             "block_tables"})
    .Outputs({"kv_cache_out"})
    .SetInplaceMap({{"kv_cache", "kv_cache_out"}})
    .Attrs({"cache_quant_type_str: std::string",
            "max_seq_len: int",
            "speculate_decoder: bool"})
    .SetKernelFn(PD_KERNEL(DecodeMLAWriteCacheKernel));